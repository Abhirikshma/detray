#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/cuda_defs.hpp"
#include "navigator_cuda_kernel.hpp"

namespace detray {

__global__ void navigator_test_kernel(
    navigator_view<navigator_host_t> n_data,
    vecmem::data::vector_view<intersection> candidates_data,
    vecmem::data::vector_view<track<nav_context>> tracks_data) {

    vecmem::device_vector<track<nav_context>> tracks(tracks_data);

    auto& traj = tracks[threadIdx.x];

    navigator_device_t n(n_data);
    navigator_device_t::state state(candidates_data);

    auto& detector = n.get_detector();

    // Set initial volume
    state.set_volume(0u);

    // Start propagation and record volume IDs
    bool heartbeat = n.status(state, traj);

    while (heartbeat) {
        heartbeat = n.target(state, traj);

        traj.pos = traj.pos + state() * traj.dir;

        heartbeat = n.status(state, traj);

        // printf("%lu \n", state.volume());
    }
}

void navigator_test(
    navigator_view<navigator_host_t> n_data,
    vecmem::data::vector_view<intersection>& candidates_data,
    vecmem::data::vector_view<track<nav_context>>& tracks_data) {

    constexpr int block_dim = 1;
    constexpr int thread_dim = 1;

    // run the test kernel
    navigator_test_kernel<<<block_dim, thread_dim>>>(n_data, candidates_data,
                                                     tracks_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray