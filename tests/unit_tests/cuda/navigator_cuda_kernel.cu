#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/cuda_defs.hpp"
#include "navigator_cuda_kernel.hpp"

namespace detray {

__global__ void navigator_test_kernel(
    navigator_view<navigator_host_t> n_data,
    vecmem::data::vector_view<intersection> candidates_data,
    const track<nav_context> traj) {

    navigator_device_t n(n_data);
    navigator_device_t::state state(candidates_data);

    auto& detector = n.get_detector();

    // Set initial volume (no grid yet)
    state.set_volume(0u);

    // Initial status call
    bool heartbeat = n.status(state, traj);

    // Let's immediately target, nothing should change, as there is full trust
    // heartbeat = n.target(state, traj);
}

void navigator_test(navigator_view<navigator_host_t> n_data,
                    vecmem::data::vector_view<intersection>& candidates_data,
                    const track<nav_context>& track) {

    constexpr int block_dim = 1;
    constexpr int thread_dim = 1;

    // run the test kernel
    navigator_test_kernel<<<block_dim, thread_dim>>>(n_data, candidates_data,
                                                     track);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray