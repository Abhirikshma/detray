#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <vecmem/containers/device_vector.hpp>

#include "definitions/cuda_defs.hpp"
#include "transform_store_cuda_kernel.hpp"

namespace detray {

    __global__ void transform_test_kernel(static_transform_store_data store_data){

	//static_transform_store<vecmem::device_vector>::context ctx0;
	static_transform_store<vecmem::device_vector> store(store_data);
	

    }
    
    void transform_test(static_transform_store_data& store_data){

	int block_dim = 1;
	int thread_dim(store_data._data.size());
	
	// run the kernel
	transform_test_kernel<<<block_dim, thread_dim>>>(store_data);
	
	// cuda error check
	DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
	DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
    }
}
